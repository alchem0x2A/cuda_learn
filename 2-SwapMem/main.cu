
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>


// The swap of pointer should not be done at the device side,
// rather the host side!
// Only pass by reference works
template<typename T>
void cudaSwap(T *&a, T *&b){
  // Swap the memory if pointers a and b
  T *tmp = a;
  a = b;
  b = tmp;
}



int main(int argc, char* argv[]){
  float *a = new float[10];
  float *b = new float[10];
  std::fill(a, &a[10], 1.0f);
  std::fill(b, &b[10], 2.0f);

  float *da, *db;
  hipMalloc((void **) &da, 10 * sizeof(float));
  hipMalloc((void **) &db, 10 * sizeof(float));

  std::cout << a[0] << " " << b[0] << std::endl;
  hipMemcpy(da, a, 10 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(db, b, 10 * sizeof(float), hipMemcpyHostToDevice);

  // both works
  // cudaSwap(da, db);
  std::swap(da, db);

  hipMemcpy(a, da, 10 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b, db, 10 * sizeof(float), hipMemcpyDeviceToHost);

  std::cout << a[0] << " " << b[0] << std::endl;
  delete a; delete b;
  hipFree(da); hipFree(db);
  
  return 0;
}
